// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
  printf("11. CUDA Runtime API Unions synthetic test\n");

#if CUDA_VERSION >= 11000
  // CHECK: hipKernelNodeAttrValue KernelNodeAttrValue;
  hipKernelNodeAttrValue KernelNodeAttrValue;
#endif

#if CUDA_VERSION >= 11080
  // CHECK: hipLaunchAttributeValue LaunchAttributeValue;
  // CHECK-NEXT: hipLaunchAttributeValue launchAttributeValue_union;
  hipLaunchAttributeValue LaunchAttributeValue;
  hipLaunchAttributeValue launchAttributeValue_union;
#endif

  return 0;
}
